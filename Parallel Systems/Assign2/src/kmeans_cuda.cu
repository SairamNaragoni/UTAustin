#include "hip/hip_runtime.h"
#include <kmeans_cuda.h>

using namespace std;

extern bool debug;
extern bool timer_debug;

__global__ void compute_distances(int num_clusters, int dims, int num_points, real *d_points, real *centroids, real *point_to_centroid_distances) {
    const unsigned int point_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (point_idx < num_points){
        const unsigned int c_idx = threadIdx.y;
        int dim_idx = threadIdx.z;

        real point = d_points[point_idx * dims + dim_idx];
        real centroid = centroids[c_idx * dims + dim_idx];
        real diff = point-centroid;
        diff = diff * diff;
        atomicAdd(&point_to_centroid_distances[point_idx * num_clusters + c_idx], diff);
    }
}

__global__ void compute_distances_shared(int num_clusters, int dims, int num_points, real *d_points, real *centroids, real *point_to_centroid_distances) {
    extern __shared__ real shared_centroids[];

    const unsigned int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int c_idx = threadIdx.y;

    if (c_idx < num_clusters) {
        // Load centroids into shared memory
        for (int dim_idx = 0 ; dim_idx < dims; dim_idx++ ){
            shared_centroids[c_idx * dims + dim_idx] = centroids[c_idx * dims + dim_idx];
        }
    }
    __syncthreads();  // Synchronize to ensure all threads have loaded data
    

    if (point_idx < num_points) {
        real diff = 0.0;
        // Loop over all dimensions to compute the distance
        for (int dim_idx = 0; dim_idx < dims; dim_idx++) {
            real point = d_points[point_idx * dims + dim_idx];
            real centroid = shared_centroids[c_idx * dims + dim_idx];
            diff += (point - centroid) * (point - centroid);
        }
        // Store the computed distance
        point_to_centroid_distances[point_idx * num_clusters + c_idx] = diff;
    }
}

__global__ void assign_points_to_clusters(int num_clusters, int dims, int num_points, real *d_points, real *point_to_centroid_distances, unsigned int* d_cluster_sizes, int *d_cluster_id_of_points) {
    const unsigned int point_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (point_idx < num_points){
        int best_centroid = -1;
        real best_centroid_dist = INFINITY;

        for (int i = 0; i < num_clusters; i++) {
            real distance = point_to_centroid_distances[point_idx*num_clusters + i];
            if (best_centroid_dist > distance) {
                best_centroid_dist = distance;
                best_centroid = i;
            }
        }
        d_cluster_id_of_points[point_idx] = best_centroid;
        atomicInc(d_cluster_sizes + best_centroid, num_points);
    }
}


__global__ void compute_new_centroids(int dims, int num_points, real *d_points, int *d_cluster_id_of_points, real *centroids) {
    const unsigned int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dim_idx = threadIdx.y;

    if (point_idx < num_points) {
        const int cluster_idx = d_cluster_id_of_points[point_idx];
        atomicAdd(&centroids[cluster_idx * dims + dim_idx], d_points[point_idx * dims + dim_idx]);
    }    
}

__global__ void compute_new_centroids_shared(int num_clusters, int dims, int num_points, real *d_points, int *d_cluster_id_of_points, real *centroids) {
    extern __shared__ real shared_sum[];

    const unsigned int point_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Zeroing shared memory for all clusters in the block
    for (int i = threadIdx.x; i < num_clusters * dims; i += blockDim.x) {
        shared_sum[i] = 0.0;
    }
    __syncthreads();

    if (point_idx < num_points) {
        int cluster_idx = d_cluster_id_of_points[point_idx];
        
        // Add point values to shared memory for the correct cluster
        for (int dim_idx = 0; dim_idx < dims; dim_idx++) {
            atomicAdd(&shared_sum[cluster_idx * dims + dim_idx], d_points[point_idx * dims + dim_idx]);
        }
    }
    __syncthreads();

    // Writing accumulated shared memory values back to global memory
    for (int i = threadIdx.x; i < num_clusters * dims; i += blockDim.x) {
        atomicAdd(&centroids[i], shared_sum[i]);
    }
}


__global__ void compute_final_centroids(int dims, int num_clusters, unsigned int* d_cluster_sizes, real *centroids) {
    const unsigned int cluster_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dim_idx = threadIdx.y;

    if (cluster_idx < num_clusters) {
        const unsigned int cluster_size = d_cluster_sizes[cluster_idx];
        // Avoid division by zero
        if (cluster_size > 0) {
            centroids[cluster_idx * dims + dim_idx] /= cluster_size;
        }
    }
}

__global__ void converged(int num_clusters, int dims, real threshold, real *old_centroids, real *new_centroids, bool *d_converged) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (c < num_clusters*dims) {
        real diff = abs(new_centroids[c] - old_centroids[c]);
        if (diff > 10*(threshold/dims)) //x10 as per suggestion in section 3.2.3
            *d_converged = false;
    }
}


//used in alternate implementation
__global__ void assignment_changed(int num_points, int dims, real threshold, int *d_old_cluster_id_of_points, int *cluster_id_of_points, bool *d_converged) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c < num_points) {
        int diff = (cluster_id_of_points[c] - d_old_cluster_id_of_points[c]);
        if (diff != 0)
            *d_converged = false;
    }
}


int kmeans_cuda(int num_points, real *h_points, struct options_t *opts, int* cluster_id_of_points, real* h_centroids, bool use_shared_memory, double *per_iteration_time) {
    int num_threads = opts->threads;

    int dims = opts->dims;
    int num_clusters = opts->num_clusters;
    int max_num_iters = opts->max_num_iter;
    real threshold = opts->threshold;
    bool done = false;
    bool is_converged = false;
    int iterations = 0;
    bool use_alternate_convergence = opts->avoid_floating_point_convergence;

    real *d_points;
    real *d_old_centroids;
    real *d_new_centroids;
    int *d_cluster_id_of_points;
    int *d_old_cluster_id_of_points;
    real *d_point_to_centroid_distances;
    unsigned int *d_cluster_sizes;
    bool *d_converged;

    //timers
    hipEvent_t start, stop, h2d_start, h2d_stop, d2h_start, d2h_stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);

    hipEventRecord(start, 0);

    if(debug){
        cout << "dims = " << dims << endl;
        cout << "num_clusters = " << num_clusters << endl;
        cout << "max_num_iters = " << max_num_iters << endl;
        cout << "threshold = " << threshold << endl;
        cout << "num_points = " << num_points << endl;

        cout << "*********** INITIAL CENTROIDS ***********" << endl;
        print_centroids(h_centroids, num_clusters, dims);
        cout << endl;
    }

    checkCudaErrors(hipMalloc(&d_points, num_points * dims * sizeof(real)));
    checkCudaErrors(hipMalloc(&d_old_centroids, num_clusters * dims * sizeof(real)));
    checkCudaErrors(hipMalloc(&d_new_centroids, num_clusters * dims * sizeof(real)));
    checkCudaErrors(hipMalloc(&d_cluster_id_of_points, num_points * sizeof(int)));

    if (use_alternate_convergence)
        checkCudaErrors(hipMalloc(&d_old_cluster_id_of_points, num_points * sizeof(int)));
    else
        checkCudaErrors(hipMalloc(&d_old_centroids, num_clusters * dims * sizeof(real)));

    checkCudaErrors(hipMalloc(&d_point_to_centroid_distances, num_points * num_clusters * sizeof(real)));
    checkCudaErrors(hipMalloc(&d_cluster_sizes, num_clusters * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_converged, sizeof(bool)));

    hipEventRecord(h2d_start, 0);
    
    // Copy points and centroids to the device
    checkCudaErrors(hipMemcpy(d_points, h_points, num_points * dims * sizeof(real), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_cluster_id_of_points, cluster_id_of_points, num_points * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_new_centroids, h_centroids, num_clusters * dims * sizeof(real), hipMemcpyHostToDevice));

    hipEventRecord(h2d_stop, 0);
    hipEventSynchronize(h2d_stop);

    float h2d_elapsed_time = 0;
    hipEventElapsedTime(&h2d_elapsed_time, h2d_start, h2d_stop);

    // Code
    while(!done) {
        if(use_alternate_convergence)
            checkCudaErrors(hipMemcpy(d_old_cluster_id_of_points, d_cluster_id_of_points, num_points * sizeof(real), hipMemcpyDeviceToDevice));
        else
            checkCudaErrors(hipMemcpy(d_old_centroids, d_new_centroids, num_clusters * dims * sizeof(real), hipMemcpyDeviceToDevice));

        if(use_shared_memory) {
            dim3 block_dim_distances(num_threads/num_clusters, num_clusters); 
            int grid_size_distances = (num_points + block_dim_distances.x - 1) / block_dim_distances.x;
            dim3 grid_dim_distances(grid_size_distances);
            size_t shared_memory_size = block_dim_distances.y * dims * sizeof(real);
            compute_distances_shared<<<grid_dim_distances, block_dim_distances, shared_memory_size >>>(num_clusters, dims, num_points, d_points, d_new_centroids, d_point_to_centroid_distances);
        } else {
            dim3 block_dim_distances(num_threads/num_clusters/dims, num_clusters, dims); 
            int grid_size_distances = (num_points + block_dim_distances.x - 1) / block_dim_distances.x;
            dim3 grid_dim_distances(grid_size_distances);
            compute_distances<<<grid_dim_distances, block_dim_distances >>>(num_clusters, dims, num_points, d_points, d_new_centroids, d_point_to_centroid_distances);
        }

        getLastCudaError("execution of compute_distances() failed\n");

        checkCudaErrors(hipMemset(d_cluster_sizes, 0, num_clusters * sizeof(int)));
 
        dim3 block_dim_assignment(num_threads);
        int grid_size_assignment = (num_points + block_dim_assignment.x - 1) / block_dim_assignment.x;
        dim3 grid_dim_assignment(grid_size_assignment);
        assign_points_to_clusters<<<grid_dim_assignment, block_dim_assignment>>>(num_clusters, dims, num_points, d_points, d_point_to_centroid_distances, d_cluster_sizes, d_cluster_id_of_points);
        getLastCudaError("execution of assign_points_to_clusters() failed\n");

        checkCudaErrors(hipMemset(d_new_centroids, 0, num_clusters * dims * sizeof(real)));

        if(use_shared_memory) {
            dim3 block_dim_new_centroids(num_threads);
            int grid_size_new_centroids = (num_points + block_dim_new_centroids.x - 1) / block_dim_new_centroids.x;
            dim3 grid_dim_new_centroids(grid_size_new_centroids);
            compute_new_centroids_shared<<<grid_dim_new_centroids, block_dim_new_centroids, num_clusters * dims * sizeof(real)>>>(num_clusters, dims, num_points, d_points, d_cluster_id_of_points, d_new_centroids);
        } else {
            dim3 block_dim_new_centroids(num_threads/dims, dims);
            int grid_size_new_centroids = (num_points + block_dim_new_centroids.x - 1) / block_dim_new_centroids.x;
            dim3 grid_dim_new_centroids(grid_size_new_centroids);
            compute_new_centroids<<<grid_dim_new_centroids, block_dim_new_centroids>>>(dims, num_points, d_points, d_cluster_id_of_points, d_new_centroids);
        }
        getLastCudaError("execution of compute_new_centroids() failed\n");

        //to reduce floating point error accumulations
        dim3 block_dim_final_centroids(num_threads/dims, dims);
        int grid_size_final_centroids = (num_clusters + block_dim_final_centroids.x - 1) / block_dim_final_centroids.x;
        dim3 grid_dim_final_centroids(grid_size_final_centroids);
        compute_final_centroids<<<grid_dim_final_centroids, block_dim_final_centroids>>>(dims, num_points, d_cluster_sizes, d_new_centroids);
        getLastCudaError("execution of compute_final_centroids() failed\n");

        checkCudaErrors(hipMemset(d_converged, 1, sizeof(bool)));

        if (use_alternate_convergence) {
            dim3 block_dim_converged(num_threads);
            int grid_size_converged = (num_points + block_dim_converged.x - 1) / block_dim_converged.x;
            dim3 grid_dim_converged(grid_size_converged);
            assignment_changed<<<grid_dim_converged, block_dim_converged>>>(num_points, dims, threshold, d_old_cluster_id_of_points, d_cluster_id_of_points, d_converged);
        } else {
            dim3 block_dim_converged(num_threads);
            int grid_size_converged = (num_clusters * dims + block_dim_converged.x - 1) / block_dim_converged.x;
            dim3 grid_dim_converged(grid_size_converged);
            converged<<<grid_dim_converged, block_dim_converged>>>(num_clusters, dims, threshold, d_old_centroids, d_new_centroids, d_converged);
        }
        getLastCudaError("execution of converged() failed\n");

        checkCudaErrors(hipMemcpy(&is_converged, d_converged, sizeof(is_converged), hipMemcpyDeviceToHost));
        iterations++;
        done = (iterations > max_num_iters) || is_converged;

        if(debug){
            cout << "*********** CENTROIDS " << iterations << " ***********" << endl;
            checkCudaErrors(hipMemcpy(h_centroids, d_new_centroids, num_clusters * dims * sizeof(real), hipMemcpyDeviceToHost));
            print_centroids(h_centroids, num_clusters, dims);
        }
    }

    hipEventRecord(d2h_start, 0);

    // Copy results back to host
    checkCudaErrors(hipMemcpy(h_centroids, d_new_centroids, num_clusters * dims * sizeof(real), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cluster_id_of_points, d_cluster_id_of_points, num_points * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(d2h_stop, 0);
    hipEventSynchronize(d2h_stop);

    float d2h_elapsed_time = 0;
    hipEventElapsedTime(&d2h_elapsed_time, d2h_start, d2h_stop);

    checkCudaErrors(hipFree(d_points));
    checkCudaErrors(hipFree(d_new_centroids));
    if(use_alternate_convergence)
        checkCudaErrors(hipFree(d_old_cluster_id_of_points));
    else
        checkCudaErrors(hipFree(d_old_centroids));
    checkCudaErrors(hipFree(d_cluster_id_of_points));
    checkCudaErrors(hipFree(d_point_to_centroid_distances));
    checkCudaErrors(hipFree(d_cluster_sizes));
    checkCudaErrors(hipFree(d_converged));

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    *per_iteration_time = elapsed_time/iterations;

    if(timer_debug) {
        float io_time = h2d_elapsed_time + d2h_elapsed_time;
        printf("cuda_per_iteration: %f ms \n", *per_iteration_time);
        printf("cuda_elapsed_time: %f ms \n", elapsed_time);
        printf("cuda_io_time: %f ms \n", io_time);
        printf("cuda_percent_spent_in_io: %f \n", io_time / elapsed_time );
    }

    return iterations;
}