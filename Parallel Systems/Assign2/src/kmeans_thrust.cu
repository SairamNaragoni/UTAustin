#include "hip/hip_runtime.h"
#include "kmeans_thrust.h"

using namespace std;

extern bool debug;
extern bool timer_debug;

template <typename T>
void print_thrust_real(thrust::device_vector<T> vec) {
    cout << "Size=" << vec.size() << endl;
    for (int i=0 ; i < vec.size(); i++){
        cout << vec[i] << " ";
    }
    cout << endl;
}

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

struct euclidean_distance_functor : public thrust::unary_function< int, real > {
    int num_clusters;
    int dims;
    real *d_centroids;
    real *d_points;

    euclidean_distance_functor(int _num_clusters, int _dims, real *_d_centroids, real *_d_points) 
        : num_clusters(_num_clusters), dims(_dims), d_centroids(_d_centroids), d_points(_d_points) {}
    
    __device__
    real operator()(int idx) {
        int point_idx = (idx / dims) / num_clusters;
        int cluster_idx = (idx / dims) % num_clusters;
        int dim_idx = idx % dims;
        real diff = d_points[point_idx * dims + dim_idx] - d_centroids[cluster_idx * dims + dim_idx];
        return diff * diff;
    }
};

struct assign_clusters : public thrust::unary_function< int, int > {
    int num_clusters;
    real *d_point_to_centroid_distances;

    assign_clusters(int _num_clusters, real *d_point_to_centroid_distances) 
        : num_clusters(_num_clusters), d_point_to_centroid_distances(d_point_to_centroid_distances) {}
    
    __device__
    real operator()(int idx) {
        real best_distance = INFINITY;
        int best_centroid = -1;
        for (int c=0; c<num_clusters; c++) {
            real distance = d_point_to_centroid_distances[idx * num_clusters + c];
            if (distance < best_distance ){
                best_distance = distance;
                best_centroid = c;
            }
        }
        return best_centroid;
    }
};

struct compute_new_centroids : public thrust::unary_function<void, thrust::tuple<real, int>> {
    int dims;
    int *d_cluster_id_of_points;
    real *centroids;

    compute_new_centroids(int _dims, int *_d_point_cluster_ids, real *_centroids)
        : dims(_dims), d_cluster_id_of_points(_d_point_cluster_ids), centroids(_centroids) {}

    __device__
    void operator()(thrust::tuple<real, int> point_idx_tuple) {
        real point_value = thrust::get<0>(point_idx_tuple);
        int idx = thrust::get<1>(point_idx_tuple);
        int assigned_centroid = d_cluster_id_of_points[idx / dims];
        int centroid_idx = assigned_centroid*dims + idx % dims;
        atomicAdd(&centroids[centroid_idx], point_value);
    }
};

struct compute_cluster_sizes : public thrust::unary_function<void, int> {
    int num_points;
    int *d_cluster_sizes;

    compute_cluster_sizes(int _num_points, int *_d_cluster_sizes)
        : num_points(_num_points), d_cluster_sizes(_d_cluster_sizes) {}

    __device__
    void operator()(int value) {
        atomicAdd(&d_cluster_sizes[value], 1);
    }
};

struct divide_centroids : public thrust::unary_function<real, thrust::tuple<int, int>> {
    int dims;
    int *d_cluster_sizes;

    divide_centroids(int _dims, int *_d_cluster_sizes)
        : dims(_dims), d_cluster_sizes(_d_cluster_sizes) {}

    __device__
    real operator()(thrust::tuple<real, int> centroid_idx_tuple) {
        real summed_centroid_value = thrust::get<0>(centroid_idx_tuple);
        int idx = thrust::get<1>(centroid_idx_tuple);
        int d_cluster_sizes_idx = idx / dims;
        int cluster_size = d_cluster_sizes[d_cluster_sizes_idx];
        if (cluster_size > 0)
            return summed_centroid_value/d_cluster_sizes[d_cluster_sizes_idx];
        else
            return summed_centroid_value;
    }
};

struct converged : public thrust::unary_function<bool, thrust::tuple<real, real>>{
    real threshold;
    int dims;

    converged(real _threshold, int _dims) : threshold(_threshold), dims(_dims) {}

    __device__
    bool operator()(thrust::tuple<real, real> centroid_pair) {
        real new_centroid = thrust::get<0>(centroid_pair);
        real old_centroid = thrust::get<1>(centroid_pair);
        real diff = abs(new_centroid - old_centroid);
        return diff < 10 * (threshold/dims);  // Return true if difference is within threshold, x10 as per suggestion in section 3.2.3
    }
};

struct assignment_changed : public thrust::unary_function<bool, thrust::tuple<int, int>> {

    __device__
    bool operator()(thrust::tuple<int, int> cluster_id_pair) {
        real old_cluster_id = thrust::get<0>(cluster_id_pair);
        real new_cluster_id = thrust::get<1>(cluster_id_pair);
        real diff = (old_cluster_id - new_cluster_id);
        return diff == 0;  // Return true if difference is 0
    }
};


int kmeans_thrust(int num_points, real *points, struct options_t *opts, int* cluster_id_of_points, real* centroids, double *per_iteration_time) {
    int dims = opts->dims;
    int num_clusters = opts->num_clusters;
    int max_num_iters = opts->max_num_iter;
    real threshold = opts->threshold;
    bool done = false;
    int iterations = 0;
    bool use_alternate_convergence = opts->avoid_floating_point_convergence;

    //timers
    hipEvent_t start, stop, h2d_start, h2d_stop, d2h_start, d2h_stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);

    hipEventRecord(start, 0);

    thrust::device_vector<int> d_cluster_id_of_points(num_points);
    thrust::device_vector<int> d_old_cluster_id_of_points(num_points);
    thrust::device_vector<int> d_cluster_sizes(num_clusters, 0);
    thrust::device_vector<real> d_new_centroids(num_clusters * dims, 0.0f);
    thrust::device_vector<real> d_points(num_points * dims);
    thrust::device_vector<real> d_old_centroids(num_clusters * dims, 0.0f);
    thrust::device_vector<real> d_point_to_centroid_distances(num_points * num_clusters);

    hipEventRecord(h2d_start, 0);

    // Copy points from host to device
    thrust::copy(points, points + (num_points * dims), d_points.begin());
    // Copy centroids from host to device
    thrust::copy(centroids, centroids + (num_clusters * dims), d_new_centroids.begin());
    // Copy cluster IDs from host to device
    thrust::copy(cluster_id_of_points, cluster_id_of_points + num_points, d_cluster_id_of_points.begin());

    hipEventRecord(h2d_stop, 0);
    hipEventSynchronize(h2d_stop);

    float h2d_elapsed_time = 0;
    hipEventElapsedTime(&h2d_elapsed_time, h2d_start, h2d_stop);

    if(debug){
        cout << "dims = " << dims << endl;
        cout << "num_clusters = " << num_clusters << endl;
        cout << "max_num_iters = " << max_num_iters << endl;
        cout << "threshold = " << threshold << endl;
        cout << "num_points = " << num_points << endl;

        cout << "*********** INITIAL CENTROIDS ***********" << endl;
        // print_centroids(centroids, num_clusters, dims);
        // print_thrust_real(d_new_centroids);
        cout << endl;
    }

    while(!done) {
        if(use_alternate_convergence)
            thrust::copy(d_cluster_id_of_points.begin(), d_cluster_id_of_points.end(), d_old_cluster_id_of_points.begin());
        else
            thrust::copy(d_new_centroids.begin(), d_new_centroids.end(), d_old_centroids.begin());

        euclidean_distance_functor euclidean_distance_functor(
            num_clusters, 
            dims, 
            raw_pointer_cast(d_new_centroids.data()), 
            raw_pointer_cast(d_points.data())
        );

        // Step 1: Compute distances between points and centroids
        thrust::reduce_by_key(
            thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(dims)),
            thrust::make_transform_iterator(thrust::counting_iterator<int>(num_points * num_clusters * dims), linear_index_to_row_index<int>(dims)),
            thrust::make_transform_iterator(thrust::counting_iterator<int>(0), euclidean_distance_functor),
            thrust::make_discard_iterator(),
            d_point_to_centroid_distances.begin()
        );

        // Step 2: Assign each point to the closest centroid
        assign_clusters assign_clusters(num_clusters, raw_pointer_cast(d_point_to_centroid_distances.data()));
        
        thrust::transform (
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(num_points),
            d_cluster_id_of_points.begin(),
            assign_clusters
        );

        // Step 3: Update centroids
        thrust::fill(d_new_centroids.begin(), d_new_centroids.end(), 0.0f);

        compute_new_centroids compute_new_centroids(
            dims, 
            raw_pointer_cast(d_cluster_id_of_points.data()), 
            raw_pointer_cast(d_new_centroids.data())
        );

        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    d_points.begin(), 
                    thrust::counting_iterator<int>(0)
                )
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    d_points.end(), 
                    thrust::counting_iterator<int>(num_points * dims)
                )
            ),
            compute_new_centroids
        );

        thrust::fill(d_cluster_sizes.begin(), d_cluster_sizes.end(), 0);
        thrust::for_each(
            d_cluster_id_of_points.begin(),
            d_cluster_id_of_points.end(),
            compute_cluster_sizes(num_clusters, raw_pointer_cast(d_cluster_sizes.data()))
        );

        thrust::transform(
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    d_new_centroids.begin(), 
                    thrust::counting_iterator<int>(0)
                )
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    d_new_centroids.end(), 
                    thrust::counting_iterator<int>(num_clusters * dims)
                )
            ),
            d_new_centroids.begin(),
            divide_centroids(dims, raw_pointer_cast(d_cluster_sizes.data()))
        );

        bool is_converged;

        if(use_alternate_convergence){
            is_converged = thrust::transform_reduce(
                thrust::make_zip_iterator(
                    thrust::make_tuple(d_old_cluster_id_of_points.begin(), d_cluster_id_of_points.begin())),
                thrust::make_zip_iterator(
                    thrust::make_tuple(d_old_cluster_id_of_points.end(), d_cluster_id_of_points.end())),
                assignment_changed(),         // Transformation: Compare distance
                true,                     // Initial value for reduction (assume converged)
                thrust::logical_and<bool>()  // Reduction: Return true only if all elements meet the condition
            );
        }
        else{
            is_converged = thrust::transform_reduce(
                thrust::make_zip_iterator(
                    thrust::make_tuple(d_new_centroids.begin(), d_old_centroids.begin())),
                thrust::make_zip_iterator(
                    thrust::make_tuple(d_new_centroids.end(), d_old_centroids.end())),
                converged(threshold, dims),         // Transformation: Compare distance
                true,                     // Initial value for reduction (assume converged)
                thrust::logical_and<bool>()  // Reduction: Return true only if all elements meet the condition
            );
        }

        iterations ++;
        done = (iterations > max_num_iters) || is_converged;
    }

    hipEventRecord(d2h_start, 0);

    thrust::copy(d_new_centroids.begin(), d_new_centroids.end(), centroids);
    thrust::copy(d_cluster_id_of_points.begin(), d_cluster_id_of_points.end(), cluster_id_of_points);

    hipEventRecord(d2h_stop, 0);
    hipEventSynchronize(d2h_stop);

    float d2h_elapsed_time = 0;
    hipEventElapsedTime(&d2h_elapsed_time, d2h_start, d2h_stop);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    *per_iteration_time = elapsed_time/iterations;

    if(timer_debug) {
        float io_time = h2d_elapsed_time + d2h_elapsed_time;
        printf("thrust_per_iteration: %f ms \n", *per_iteration_time);
        printf("thrust_elapsed_time: %f ms \n", elapsed_time);
        printf("thrust_io_time: %f ms \n", io_time);
        printf("thrust_percent_spent_in_io: %f \n", io_time / elapsed_time );
    }

    return iterations;
}